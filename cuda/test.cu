
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define ARR_SIZE 100000000
#define BLOCK_SIZE 256
__global__ void balls(int a[], int b[], int c[]) { int i = blockIdx.x * blockDim.x * threadIdx.x; c[i] = a[i] + b[i]; }


int main(int argc, char **argv)
{
	srand(time(NULL));
	int *a = (int *)malloc(ARR_SIZE * sizeof(int));
	int *b = (int *)malloc(ARR_SIZE * sizeof(int));
	int *c = (int *)malloc(ARR_SIZE * sizeof(int));
	for(int i = 0;i < ARR_SIZE; i++)
	{
		a[i] = rand() /  10000;
		b[i] = rand() /  10000;
	}
	int *dev_a;
	int *dev_b;
	int *dev_c;
	
	hipMalloc((void **)&dev_a, sizeof(int) * ARR_SIZE);
	hipMalloc((void **)&dev_b, sizeof(int) * ARR_SIZE);
	hipMalloc((void **)&dev_c, sizeof(int) * ARR_SIZE);

	hipMemcpy(dev_a, a, sizeof(int) * ARR_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(int) * ARR_SIZE, hipMemcpyHostToDevice);
	int numBlocks = (ARR_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
	balls<<<numBlocks, BLOCK_SIZE>>>(dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, sizeof(int) * ARR_SIZE , hipMemcpyDeviceToHost);
	// printf("%d + %d = %d\n", a[0], b[0], c[0]);
	// printf("%d + %d = %d\n", a[1], b[1], c[1]);
	// printf("%d + %d = %d\n", a[2], b[2], c[2]);
	// printf("%d + %d = %d\n", a[3], b[3], c[3]);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}
